#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include "timer.h"

#define SOFTENING 1e-4f
#define MAX_BLOCKS_PER_DIM 65535
#define MIN(a, b) (((a) < (b)) ? (a) : (b))

struct Body
{
    float x, y, z, vx, vy, vz;
};
typedef struct Body Body;

__host__ void particle_positions_to_csv(FILE *datafile, int iter, Body *p, int nBodies)
{
    for (int i = 0; i < nBodies; i++)
    {
        fprintf(datafile, "%i, %f, %f, %f\n", iter, p[i].x, p[i].y, p[i].z);
    }
}

__host__ void randomizeBodies(Body *data, int n)
{
    for (int i = 0; i < n / 2; i++)
    {
        data[i].x = 0.0 + (rand() / (float)RAND_MAX) * 100;
        data[i].y = 100.0 + (rand() / (float)RAND_MAX) * 100;
        data[i].z = 0.0 + (rand() / (float)RAND_MAX) * 100;

        data[i + n / 2].x = -100.0 + (rand() / (float)RAND_MAX) * 100;
        data[i + n / 2].y = 0.0 + (rand() / (float)RAND_MAX) * 100;
        data[i + n / 2].z = 0.0 + (rand() / (float)RAND_MAX) * 100;

        // data[i].vx = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
        // data[i].vy = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
        // data[i].vz = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;

        // data[i + n / 2].vx = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
        // data[i + n / 2].vy = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
        // data[i + n / 2].vz = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;

        data[i].vx = 0;
        data[i].vy = 0;
        data[i].vz = 0;

        data[i + n / 2].vx = 0;
        data[i + n / 2].vy = 0;
        data[i + n / 2].vz = 0;
    }
}
__global__ void test(Body *particles, int nBodies,
                     float Fx, float Fy, float Fz, int i)
{

    for (int j = 0; j < nBodies; j++)
    {
        float dx = particles[j].x - particles[i].x;
        float dy = particles[j].y - particles[i].y;
        float dz = particles[j].z - particles[i].z;
        float distSqr = dx * dx + dy * dy + dz * dz + SOFTENING;
        float invDist = 1.0f / sqrtf(distSqr);
        float invDist3 = invDist * invDist * invDist;

        Fx += dx * invDist3;
        Fy += dy * invDist3;
        Fz += dz * invDist3;
    }
}
__global__ void bodyForce(Body *particles, float dt, int nBodies)
{
    int tid0 = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = tid0; i < nBodies; i += blockDim.x * gridDim.x)
    {
        float Fx = 0.0f;
        float Fy = 0.0f;
        float Fz = 0.0f;

        for (int j = 0; j < nBodies; j++)
        {
            float dx = particles[j].x - particles[i].x;
            float dy = particles[j].y - particles[i].y;
            float dz = particles[j].z - particles[i].z;
            float distSqr = dx * dx + dy * dy + dz * dz + SOFTENING;
            float invDist = 1.0f / sqrtf(distSqr);
            float invDist3 = invDist * invDist * invDist;

            Fx += dx * invDist3;
            Fy += dy * invDist3;
            Fz += dz * invDist3;
        }

        particles[i].vx += dt * Fx;
        particles[i].vy += dt * Fy;
        particles[i].vz += dt * Fz;
    }

    // INTEGRATE POSITION
    for (int i = tid0; i < nBodies; i += blockDim.x * gridDim.x)
    {
        particles[i].x += particles[i].vx * dt;
        particles[i].y += particles[i].vy * dt;
        particles[i].z += particles[i].vz * dt;
    }
}

int main(const int argc, const char **argv)
{
    int nthreads_per_block = 128, nblocks;
    int nBodies = 3000;

    if (argc > 1)
        nBodies = atoi(argv[1]);
    if (argc > 2)
        nthreads_per_block = atoi(argv[2]);

    nblocks = MIN(nBodies / nthreads_per_block + 1, MAX_BLOCKS_PER_DIM);

    Body *particles_h = (Body *)malloc(sizeof(Body) * nBodies);
    Body *particles_d;
    assert(hipMalloc((void **)&particles_d, sizeof(Body) * nBodies) == hipSuccess);

    randomizeBodies(particles_h, nBodies); // Init pos / vel data
    hipMemcpy(particles_d, particles_h, sizeof(Body) * nBodies, hipMemcpyHostToDevice);

    // TIME STEP
    const float dt = 0.01f;
    // SIMULATION ITERATIONS
    const int nIters = 20;

#ifdef WRITETOFILE
    FILE *datafile = fopen("nbody.csv", "w");
    particle_positions_to_csv(datafile, 0, particles_h, nBodies);
#endif
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    {
        // ITERATE
        for (int iter = 1; iter <= nIters; iter++)
        {
            // printf("iteration:%d\n", iter);

            // COMPUTE INTERBODY FORCES
            bodyForce<<<nblocks, nthreads_per_block>>>(particles_d, dt, nBodies);

#ifdef WRITETOFILE
            hipMemcpy(particles_h, particles_d, sizeof(Body) * nBodies, hipMemcpyDeviceToHost);

            particle_positions_to_csv(datafile, iter, particles_h, nBodies);
#endif
        }
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float totalTime = 0.0;
    hipEventElapsedTime(&totalTime, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    free(particles_h);
    hipFree(particles_d);

    totalTime /= (float)1000;
    double avgTime = totalTime / (float)(nIters);
    printf("avgTime: %f (s)   totTime: %f (s)\n", avgTime, totalTime);

#ifdef WRITETOFILE
    fclose(datafile);
#endif
    return EXIT_SUCCESS;
}
